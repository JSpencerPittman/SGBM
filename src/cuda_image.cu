#include "cuda_image.cuh"

CudaImage::CudaImage(Image& image):
    m_path(image.path()), m_isGrayscale(image.isGrayscale()),
    m_width(image.width()), m_height(image.height()),
    m_channels(image.channels()) {
        hipMalloc(&m_data, image.size());
        hipMemcpy(m_data, image.data(), image.size(), hipMemcpyHostToDevice);
}

CudaImage::~CudaImage() { hipFree(m_data); }
